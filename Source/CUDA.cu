#include "hip/hip_runtime.h"
// CUDA.cu - CORVID CHESS ENGINE (c) 2019 Stuart Riffle

#include "Core.h"
#include "Job.h"
#include "CUDA.h"

__global__ void PlayGamesOnDevice( const PlayoutJobInfo* input, PlayoutJobresult* result, int count )
{
    GamePlayer player( &input->mOptions );

    result.mScores = player.PlayGames( input->mPos, count );
    result.mPathFromRoot = input->mPathFromRoot;
}

void QueuePlayoutJobCuda( CudaJob* job, int blockCount, int blockSize )
{
    // Copy the inputs to device

    hipMemcpyAsync( 
        job->mInputDev, 
        job->mInputHost, 
        sizeof( PlayoutJobInfo ), 
        hipMemcpyHostToDevice, 
        job->mStream );

    // Queue the playout kernel

    hipEventRecord( job->mStartEvent, job->mStream );

    PlayGamesOnDevice<<< blockCount, blockSize, 0, job->mStream >>>( 
        job->mInputDev, 
        job->mOutputDev, 
        job->mCount );

    hipEventRecord( job->mEndEvent, job->mStream );

    // Copy the results back to host

    hipMemcpyAsync( 
        job->mOutputHost, 
        job->mOutputDev, 
        sizeof( PlayoutJobResult ), 
        hipMemcpyDeviceToHost, 
        job->mStream );

    // Record an event we can test for completion

    hipEventRecord( job->mReadyEvent, job->mStream );
}

