#include "hip/hip_runtime.h"
// GPU-CUDA.cu - CORVID CHESS ENGINE (c) 2012-2016 Stuart Riffle

#include "Core.h"

__global__ void PlayGamesOnDevice( const PlayoutJobInput* input, PlayoutJobOutput* output, int count )
{
    PlayoutProvider player( &input->mOptions );

    output->mScores = player.PlayGames( input->mPos, count );
    output->mPathFromRoot = input->mPathFromRoot;
}

void QueuePlayoutJobCuda( PlayoutJobInput* batch, int blockCount, int blockSize, i32* exitFlag )
{
    // Copy the inputs to device

    hipMemcpyAsync( 
        batch->mInputDev, 
        batch->mInputHost, 
        sizeof( PlayoutJobInput ) * batch->mCount, 
        hipMemcpyHostToDevice, 
        batch->mStream );

    // Clear the device outputs

    hipMemsetAsync( 
        batch->mOutputDev, 
        0, 
        sizeof( PlayoutJobOutput ) * batch->mCount, 
        batch->mStream );

    // Run the search kernel

    hipEventRecord( batch->mStartEvent, batch->mStream );

    SearchPositionsOnGPU<<< blockCount, blockSize, 0, batch->mStream >>>( 
        batch->mInputDev, 
        batch->mOutputDev, 
        batch->mCount, 
        stride, 
        batch->mHashTableDev, 
        batch->mEvaluatorDev,
        batch->mOptionsDev,
        exitFlag );

    hipEventRecord( batch->mEndEvent, batch->mStream );

    // Copy the outputs to host

    hipMemcpyAsync( batch->mOutputHost, batch->mOutputDev, sizeof( PlayoutJobOutput ) * batch->mCount, hipMemcpyDeviceToHost, batch->mStream );

    // Record an event we can test for completion

    hipEventRecord( batch->mReadyEvent, batch->mStream );
}

