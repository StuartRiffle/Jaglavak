#include "hip/hip_runtime.h"
// JAGLAVAK CHESS ENGINE (c) 2019 Stuart Riffle

#include "Platform.h"
#include "Chess.h"
#include "PlayoutBatch.h"
#include "GamePlayer.h"

__global__ void PlayGamesCuda( const PlayoutParams* params, const Position* pos, ScoreCard* dest, int count )
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    GamePlayer player( params, x );
    
    int idx = x % count;

    ScoreCard scores;
    player.PlayGames( pos + idx, &scores, 1 );

    atomicAdd( (unsigned long long*) &dest[idx].mWins[BLACK], scores.mWins[BLACK] );
    atomicAdd( (unsigned long long*) &dest[idx].mWins[WHITE], scores.mWins[WHITE] );
    atomicAdd( (unsigned long long*) &dest[idx].mPlays, scores.mPlays );
}

void PlayGamesCudaAsync( CudaLaunchSlot* slot, int blockCount, int blockSize, hipStream_t stream )
{
    PlayGamesCuda<<< blockCount, blockSize, 0, stream >>>(
        slot->mParams.mDev,
        slot->mInputs.mDev, 
        slot->mOutputs.mDev, 
        slot->mCount );
}
