#include "hip/hip_runtime.h"
// PlayoutKernel.cu - CORVID CHESS ENGINE (c) 2019 Stuart Riffle

#include "Core.h"
#include "Core.h"
#include "PlayoutJob.h"


__global__ void PlayGamesCuda( const PlayoutJob* job, PlayoutResult* result, int count )
{
    GamePlayer player( &job->mOptions );

    result->mScores = player.PlayGames( job->mPosition, count );
    result->mPathFromRoot = job->mPathFromRoot;
}


void QueuePlayGamesCuda( CudaLaunchSlot* slot, int blockCount, int blockSize )
{
    hipEventRecord( slot->mStartEvent, slot->mStream );

    // Copy the inputs to device

    hipMemcpyAsync( 
        slot->mInputDev, 
        slot->mInputHost, 
        sizeof( PlayoutJobInfo ), 
        hipMemcpyHostToDevice, 
        slot->mStream );

    // Queue the playout kernel

    PlayGamesOnDevice<<< blockCount, blockSize, 0, slot->mStream >>>( 
        slot->mInputDev, 
        slot->mOutputDev, 
        slot->mCount );

    // Copy the results back to host

    hipMemcpyAsync( 
        slot->mOutputHost, 
        slot->mOutputDev, 
        sizeof( PlayoutJobResult ), 
        hipMemcpyDeviceToHost, 
        slot->mStream );

    hipEventRecord( slot->mEndEvent, slot->mStream );
}

