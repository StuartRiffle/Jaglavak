#include "hip/hip_runtime.h"
// CUDA.cu - CORVID CHESS ENGINE (c) 2019 Stuart Riffle

#include "Core.h"
#include "Job.h"
#include "CUDA.h"

__global__ void PlayGamesOnDevice( const PlayoutJobInfo* input, PlayoutJobresult* result, int count )
{
    GamePlayer player( &input->mOptions );

    result->mScores = player.PlayGames( input->mPos, count );
    result->mPathFromRoot = input->mPathFromRoot;
}


void QueuePlayoutJobCuda( CudaLaunchSlot* slot, int blockCount, int blockSize )
{
    hipEventRecord( slot->mStartEvent, slot->mStream );

    // Copy the inputs to device

    hipMemcpyAsync( 
        slot->mInputDev, 
        slot->mInputHost, 
        sizeof( PlayoutJobInfo ), 
        hipMemcpyHostToDevice, 
        slot->mStream );

    // Queue the playout kernel

    PlayGamesOnDevice<<< blockCount, blockSize, 0, slot->mStream >>>( 
        slot->mInputDev, 
        slot->mOutputDev, 
        slot->mCount );

    hipEventRecord( slot->mEndEvent, slot->mStream );

    // Copy the results back to host

    hipMemcpyAsync( 
        slot->mOutputHost, 
        slot->mOutputDev, 
        sizeof( PlayoutJobResult ), 
        hipMemcpyDeviceToHost, 
        slot->mStream );

    hipEventRecord( job->mStopEvent, job->mStream );
}

