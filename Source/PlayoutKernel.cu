#include "hip/hip_runtime.h"
// PlayoutKernel.cu - CORVID CHESS ENGINE (c) 2019 Stuart Riffle

#include "Core.h"
#include "Core.h"
#include "PlayoutJob.h"


__global__ void PlayGamesCuda( const PlayoutJob* job, PlayoutResult* result, int count )
{
    GamePlayer< u64 > player( &job->mOptions, job->mRandomSeed );

    result->mScores = player.PlayGames( job->mPosition, count );
    result->mPathFromRoot = job->mPathFromRoot;
}


void QueuePlayGamesCuda( CudaLaunchSlot* slot, int blockCount, int blockSize )
{
    hipEventRecord( slot->mStartEvent, slot->mStream );

    // Copy the inputs to device

    hipMemcpyAsync( 
        slot->mInputDev, 
        slot->mInputHost, 
        sizeof( PlayoutJob ), 
        hipMemcpyHostToDevice, 
        slot->mStream );

    // Queue the playout kernel

    PlayGamesCuda<<< blockCount, blockSize, 0, slot->mStream >>>( 
        slot->mInputDev, 
        slot->mOutputDev, 
        slot->mInfo.mNumGames );

    // Copy the results back to host

    hipMemcpyAsync( 
        slot->mOutputHost, 
        slot->mOutputDev, 
        sizeof( PlayoutResult ), 
        hipMemcpyDeviceToHost, 
        slot->mStream );

    hipEventRecord( slot->mEndEvent, slot->mStream );
}

